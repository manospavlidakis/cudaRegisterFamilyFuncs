
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>
#include <iostream>
#include <iomanip>
#include <cstddef>
#include <vector>
#include <tuple>
using namespace std;

extern hipError_t cudaGetExportTable(const void ** ppExportTable, const hipUUID * pExportTableId);
__device__ long long device_int;
__device__ int device_int2;


__global__ void kernel(long long *a)
{
    device_int= device_int2+1000;
    *a = (long long)&device_int;
    //device_int2 = (int)&device_int;
    //printf("%p:\n",&device_int);

}
void uuid_print(hipUUID a){
  std::cout << "GPU";
  std::vector<std::tuple<int, int> > r = {{0,4}, {4,6}, {6,8}, {8,10}, {10,16}};
  for (auto t : r){
    std::cout << "-";
    for (int i = std::get<0>(t); i < std::get<1>(t); i++)
      std::cout << std::hex << std::setfill('0') << std::setw(2) << (unsigned)(unsigned char)a.bytes[i];
  }
  std::cout << std::endl;
}
int main()
{
  const int * ppExportTable[1024]={0};
  printf("ppExportTable: %p\n", &ppExportTable[0]);
  //char exportTable[1024*1024*1024]; 
  const hipUUID desired_UUID = {
    (char)0xafU, 
    (char)0x3dU, 
    (char)0xd3U, //
    (char)0x92U, 
    (char)0x2aU, 
    (char)0x88U, 
    (char)0x00U, 
    (char)0x32U, 
    (char)0xcbU, 
    (char)0xccU, 
    (char)0x80U, 
    (char)0x3dU, 
    (char)0x29U, 
    (char)0x7eU, 
    (char)0x5fU, 
    (char)0xa2U};
  cudaGetExportTable(((const void**)&ppExportTable), &desired_UUID);

  uuid_print(desired_UUID);
  printf("ppExportTable: %p\n", &ppExportTable[0]);
  printf("ppExportTable[0]: %p\n", ppExportTable[0]);
  for (int i = 0; i<1024; i++){
    printf("ppExportTable[%d]: %p\n", i,ppExportTable[i]);
  }
#if 0
    long long *d_A, h_A;    
    cudaMalloc(&d_A, sizeof(long long));	
    kernel<<<1, 1>>>(d_A);
    assert(cudaGetLastError() == cudaSuccess);

    int *host_pointer_to_device_int;
    /* Get a pointer to device_int. After this, I won't be able to access it,
     * but I'm going to copy its value with cudaMemcpy */
    cudaError_t err = cudaGetSymbolAddress((void **)&host_pointer_to_device_int, "device_int");
    std::cerr<<"ptr: "<<host_pointer_to_device_int<<std::endl;
    std::cerr<<"Error: "<<cudaGetErrorString(err)<<std::endl;
    assert(cudaGetLastError() == cudaSuccess); // The above operation executed successfully
    //__cudaRegisterVar(0, 0, 0 ,0, 0, 0 ,0, 0 );
    int host_int;
    // Copy the device_int's value
    cudaMemcpy(&h_A, d_A,
            sizeof(long long), cudaMemcpyDeviceToHost);
    assert(cudaGetLastError() == cudaSuccess); // The above operation executed successfully
//    printf("host int: %lu\n",host_int );
    cout <<"A: "<<std::hex<< h_A << endl; // Everything's fine!
#endif
}
